#include "hip/hip_runtime.h"
//Paralell ant algorithm for the travelling salesman problem using Cuda
//Made by: Arthur Henrique Guimarães

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>


//Problem Parameters
#define CITIES 1000
#define ANTS 180
#define MAX_DIST 100
#define MAX_TOTAL_DISTANCE (CITIES * MAX_DIST) // MAX possible distance that an ant can walk

#define ALPHA 1
#define BETA 5       //This parameter raises the weight of distance over pheromone
#define RHO 0.5      //Evapouration rate
#define QVAL 100     //
#define MAX_TOURS 50// The number of times an ant will walk trough all the cities
#define INIT_PHER (1.0/CITIES) //Initial hormone for each path
#define BLOCKS 3
#define THREADS ANTS/BLOCKS


//Global structures
struct ant{
  // Current city, next city and total of traversed cities
  int curCity, nextCity, pathIndex;
  // List of visited cities
  int visited[CITIES];
  // Traversed path
  int path[CITIES];
  // Length of the traversed path
  float tourLength;
};

//CPU
float        distances[CITIES][CITIES]; // Distance between city i an j
double       hormone[CITIES][CITIES]; //Hormone between city i and j
struct ant   ants[ANTS];
float        bestdistance[ANTS];
float 		 finalbest = (float)MAX_TOTAL_DISTANCE;
hiprandState  state[ANTS];
const size_t distances_size = sizeof(float) * size_t(CITIES*CITIES);
const size_t hormone_size = sizeof(double) * size_t(CITIES*CITIES);

//GPU
float        *distances_d;
struct ant   *ants_d;
double       *hormone_d;
float        *bestdistance_d;
hiprandState  *state_d;

//Functions

void get_distances_matrix();
void deviceAlloc();
__global__ void initialize_ants(struct ant *ants_d, hiprandState *state_d, float *bestdistance_d);
__global__ void setup_curand_states(hiprandState *state_d, unsigned long t);
__global__ void restart_ants(struct ant *ants_d,hiprandState *state_d, float *bestdistance_d);
void move_ants();
__global__ void simulate_ants(struct ant *ants_d,hiprandState *state_d, float *distances_d, double *hormone_d);
__device__ double antProduct(int from, int to, double *hormone_d, float *distances_d);
__device__ int NextCity(struct ant *ants_d, int pos, float *distances_d, double *hormone_d, hiprandState *state_d );
void updateTrails();

int main(){

  get_distances_matrix(); // Get the distances between cities from the input
  deviceAlloc(); // Mallocs and memcpy of the device variables

  //Set up an array of curand_states in order to build better random numbers
  time_t t; time(&t);
  setup_curand_states <<< BLOCKS, THREADS >>> (state_d, (unsigned long) t);
  hipDeviceSynchronize();

  //initialize the ants array
  initialize_ants <<< BLOCKS, THREADS >>> (ants_d, state_d, bestdistance_d);
  hipDeviceSynchronize();

  // Start and control the ants tours
  move_ants();

  //Free Memory
  	hipFree(ants_d);
  	hipFree(bestdistance_d);
  	hipFree(distances_d);
  	hipFree(hormone_d);
  	hipFree(state_d);
  	hipFree(bestdistance_d);

  return 0;
}


void get_distances_matrix(){
  int i,j;
  float k;

  while(scanf("%i %i %f", &i,&j,&k) == 3){
    distances[i][j] = k;
    hormone[i][j] = INIT_PHER;
  }

}

void deviceAlloc(){
	hipMalloc( (void**) &ants_d, sizeof(ants));
	hipMalloc( (void**) &state_d, sizeof(state));

	hipMalloc( (void**) &distances_d, distances_size);
	hipMemcpy(distances_d, distances, distances_size, hipMemcpyHostToDevice);

	hipMalloc( (void**) &hormone_d, hormone_size);
	hipMemcpy(hormone_d, hormone, hormone_size, hipMemcpyHostToDevice);

	hipMalloc( (void**) &bestdistance_d, sizeof(bestdistance));
}

__global__ void setup_curand_states(hiprandState *state_d, unsigned long t){
	int id = threadIdx.x + blockIdx.x*THREADS;
	hiprand_init(t, id, 0, &state_d[id]);
}

__global__ void initialize_ants(struct ant *ants_d, hiprandState *state_d, float *bestdistance_d){

  int position = threadIdx.x + blockIdx.x*THREADS;
  int k;

  // Mark all cities as not visited
  // Mark all path as not traversed
  for(k = 0; k < CITIES; k++){
    ants_d[position].visited[k] = 0;
    ants_d[position].path[k] = -1;
  }

  bestdistance_d[position] = (float)MAX_TOTAL_DISTANCE;

  //Random City to begin
  ants_d[position].curCity = hiprand(&state_d[position])% CITIES;
  //
  ants_d[position].pathIndex = 1;
  ants_d[position].path[0] = ants_d[position].curCity;
  ants_d[position].nextCity = -1;
  ants_d[position].tourLength = 0;
  ants_d[position].visited[ants_d[position].curCity] = 1;
}

__global__ void restart_ants(struct ant *ants_d,hiprandState *state_d, float *bestdistance_d){

	int position = threadIdx.x + blockIdx.x*THREADS;
	int i;

	if(ants_d[position].tourLength < bestdistance_d[position]){
		bestdistance_d[position] = ants_d[position].tourLength;
	}

	ants_d[position].nextCity = -1;
	ants_d[position].tourLength = 0.0;

	for(i = 0; i < CITIES; i++){
		ants_d[position].visited[i] = 0;
		ants_d[position].path[i] = -1;
	}

	ants_d[position].curCity = hiprand(&state_d[position])% CITIES;
	ants_d[position].pathIndex = 1;
	ants_d[position].path[0] = ants_d[position].curCity;
	ants_d[position].visited[ants_d[position].curCity] = 1;
}

void move_ants(){
	int curtour = 0;
	while (curtour++ < MAX_TOURS){
		simulate_ants <<< BLOCKS, THREADS >>> (ants_d, state_d, distances_d, hormone_d);
		hipDeviceSynchronize();

		hipMemcpy(ants, ants_d, sizeof(ants), hipMemcpyDeviceToHost);
		updateTrails();
		hipMemcpy(hormone_d, hormone, hormone_size, hipMemcpyHostToDevice);

		int i;
				hipMemcpy(bestdistance, bestdistance_d, sizeof(bestdistance), hipMemcpyDeviceToHost);
				for(i =0; i < ANTS; i++)
				  if(bestdistance[i] < finalbest){
					  finalbest = bestdistance[i];
				  }
				printf("Best distance %f \n", finalbest);

		restart_ants <<< BLOCKS, THREADS >>> (ants_d, state_d, bestdistance_d);
		hipDeviceSynchronize();

	}
}

__global__ void simulate_ants(struct ant *ants_d,hiprandState *state_d, float *distances_d, double *hormone_d ){

	int position = threadIdx.x + blockIdx.x*THREADS;
	int curtime = 0;

	while(curtime++ < CITIES){
		if( ants_d[position].pathIndex < CITIES ){ //check if all cities were visited

			// Choose the next city to visit
			ants_d[position].nextCity = NextCity(ants_d, position, distances_d, hormone_d, state_d);
			// Mark the city as visited
			ants_d[position].visited[ants_d[position].nextCity] = 1;
			// Mark when the city was visited
			ants_d[position].path[ants_d[position].pathIndex++] = ants_d[position].nextCity;
			// Add to the tour the distance between the curCity and the NextCity
			ants_d[position].tourLength += distances_d[ants_d[position].curCity + (ants_d[position].nextCity * CITIES)];

			// Handle the last case -> path from last to first city
			if(ants_d[position].pathIndex == CITIES){
				ants_d[position].tourLength += distances_d[ants_d[position].path[CITIES -1] + (ants_d[position].path[0]*CITIES)];
			}
			// Mark NextCity as the curCity
			ants_d[position].curCity = ants_d[position].nextCity;
		}
	}

}

__device__ double antProduct(int from, int to, double *hormone_d, float *distances_d){
  return (double) (( pow( hormone_d[from + to*CITIES], ALPHA) * pow( (1.0/ distances_d[from + to*CITIES]), BETA)));
}


__device__ int NextCity(struct ant *ants_d, int pos, float *distances_d, double *hormone_d, hiprandState *state_d ){
	int to, from;
	double denom = 0.0;
	from =  ants_d[pos].curCity;

	for(to = 0; to < CITIES; to++){
	  if(ants_d[pos].visited[to] == 0){
		denom += antProduct(from, to, hormone_d, distances_d);
	  }
	}

	assert(denom != 0.0);

	to++;
	int count = CITIES - ants_d[pos].pathIndex;

	do{
		double p;
		to++;

		if(to >= CITIES)
			to = 0;

		if(ants_d[pos].visited[to] == 0){
			p = (double) antProduct(from, to, hormone_d, distances_d)/denom;
			double x = (double)(hiprand(&state_d[pos])% 1000000000000000000)/1000000000000000000;
			if(x < p){
				break;
			}
			count--;
			if(count == 0){
				break;
			}
		}
	}while(1);

	return to;
}

void updateTrails(){
	int from,to,i,ant;

	//hormone evaporation
	for(from = 0; from < CITIES; from++)
		for(to = 0;to < CITIES; to++){
			if(from!=to){
				hormone[from][to] *=( 1.0 - RHO);

				if(hormone[from][to] < 0.0){
					hormone[from][to] = INIT_PHER;
				}
			}
		}

	//add new pheromone to the trails
	for(ant = 0; ant < ANTS; ant++)
		for(i = 0; i < CITIES; i++){
			if( i < CITIES - 1 ){
				from = ants[ant].path[i];
				to = ants[ant].path[i+1];
			}
			else{
				from = ants[ant].path[i];
				to = ants[ant].path[0];
			}

			hormone[from][to] += (QVAL/ ants[ant].tourLength);
			hormone[to][from] = hormone[from][to];

		}


	for (from = 0; from < CITIES; from++)
		for( to = 0; to < CITIES; to++){
			hormone[from][to] *= RHO;
		}

}
























/*
 * //teste
	   	  int i;
	   	  hipMemcpy(ants, ants_d, sizeof(ants), hipMemcpyDeviceToHost);
	   	  for(i = 0; i < ANTS; i++)
	   	  	  printf("Ant[%i] -> curCity %i  path[curCity] = %i visited[curCity] = %i\n", i, ants[i].curCity,
	   	  			  ants[i].path[ants[i].curCity], ants[i].visited[ants[i].curCity]);
	   //teste
 *
 */
